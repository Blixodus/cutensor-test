#include <span>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#define HANDLE_ERROR(x)                                               \
{ const auto err = x;                                                 \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                \
  { printf("Error: %s\n", hiptensorGetErrorString(err)); return err; } \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{ const auto err = x;                                             \
  if( err != hipSuccess )                                        \
  { printf("Error: %s\n", hipGetErrorString(err)); return err; } \
};

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3;
    }
    private:
    hipEvent_t start_, stop_;
};

template <typename DataType, typename SizeType = size_t>
class GPUTensor {
	hiptensorHandle_t handle_;
	DataType *dataPtr_;
	std::vector<SizeType> dimSize_;
	std::vector<SizeType> ld_;
	std::vector<SizeType> dimName_;
	
public:
	GPUTensor(hiptensorHandle_t handle, DataType *dataPtr, std::span<SizeType> dimSize, std::span<SizeType> ld, std::span<SizeType> dimName)
		: handle_(handle)
	{
		
	}
	
};

int main() {
	hiptensorHandle_t handle;
	

	
	return 0;
}